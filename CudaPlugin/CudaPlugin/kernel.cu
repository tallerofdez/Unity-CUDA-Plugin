#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>
#include <iostream>
#include <cmath>
#include <string>



#pragma warning(2:4235)

#define DllExport __declspec (dllexport)

# define M_PI           3.14159265358979323846  /* pi */



__global__ void instanceParticles(float* outX, float* outY, float* outZ, float offset, int N, int WIDTH, int HEIGHT, int DEPTH) {

	unsigned int Xidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int Yidx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int Zidx = blockIdx.z * blockDim.z + threadIdx.z;
	unsigned int idx = Xidx + Yidx * WIDTH + Zidx * WIDTH * HEIGHT;

	if (Xidx < WIDTH && Yidx < HEIGHT && Zidx < DEPTH) {
		outX[idx] = Xidx * offset;
		outY[idx] = Yidx * offset;
		outZ[idx] = Zidx * offset;
	}

}

__global__ void moveParticles(float* outX, float* outY, float* outZ, float* initialX, float* initialY, float* initialZ, float movement, int N, int WIDTH, int HEIGHT, int DEPTH) {

	unsigned int Xidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int Yidx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int Zidx = blockIdx.z * blockDim.z + threadIdx.z;
	unsigned int idx = Xidx + Yidx * WIDTH + Zidx * WIDTH * HEIGHT;

	if (Xidx < WIDTH && Yidx < HEIGHT && Zidx < DEPTH) {

		outX[idx] = initialX[idx];
		outY[idx] = initialY[idx] + movement;
		outZ[idx] = initialZ[idx];

	}
}

__global__ void setInitialPos(float* inX, float* inY, float* inZ, float* outX, float* outY, float* outZ, int N, int WIDTH, int HEIGHT, int DEPTH) {
	unsigned int Xidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int Yidx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int Zidx = blockIdx.z * blockDim.z + threadIdx.z;
	unsigned int idx = Xidx + Yidx * WIDTH + Zidx * WIDTH * HEIGHT;

	if (Xidx < WIDTH && Yidx < HEIGHT && Zidx < DEPTH) {

		outX[idx] = inX[idx];
		outY[idx] = inY[idx];
		outZ[idx] = inZ[idx];

	}
}




extern "C" {



	DllExport const char*  CUDA_device_name()
	{
		hipDeviceProp_t device;
		hipGetDeviceProperties(&device, 0);
		char* label = new char[256];
		strcpy_s(label, 256, device.name);
		return label;
	}

	DllExport void Clear(float* host)
	{
		free(host);
	}

	DllExport const void  cubeFormation(float* posX, float* posY, float* posZ, int N, int cube, float offset)
	{

		const int THREAD_SIZE = cube;
		const int BLOCK_SIZE = floor(cbrt(1024));
		
		float* d_X;
		float* d_Y;
		float* d_Z;

 
		hipMalloc(&d_X, sizeof(float) * N);
		hipMalloc(&d_Y, sizeof(float) * N);
		hipMalloc(&d_Z, sizeof(float) * N);

		/**/
		dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
		dim3 gridDim(ceil((double)cube / blockDim.x), ceil((double)cube / blockDim.y), ceil((double)cube / blockDim.z));


		instanceParticles << <gridDim, blockDim >> > (d_X, d_Y, d_Z, offset, N, cube, cube, cube);
		hipDeviceSynchronize();

		hipMemcpy(posX, d_X, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipMemcpy(posY, d_Y, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipMemcpy(posZ, d_Z, sizeof(float) * N, hipMemcpyDeviceToHost);

		hipFree(d_X);
		hipFree(d_Y);
		hipFree(d_Z);

		

	}


	DllExport const void InitialPos (float* posX, float* posY, float* posZ, float* initialPosX, float* initialPosY, float* initialPosZ, int N, int cube) {
		
		
		const int THREAD_SIZE = cube;
		const int BLOCK_SIZE = floor(cbrt(1024));

		float* d_inX;
		float* d_inY;
		float* d_inZ;

		float* d_outX;
		float* d_outY;
		float* d_outZ;

		hipMalloc(&d_inX, sizeof(float) * N);
		hipMalloc(&d_inY, sizeof(float) * N);
		hipMalloc(&d_inZ, sizeof(float) * N);

		hipMalloc(&d_outX, sizeof(float) * N);
		hipMalloc(&d_outY, sizeof(float) * N);
		hipMalloc(&d_outZ, sizeof(float) * N);

		hipMemcpy(d_inX, posX, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(d_inY, posY, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(d_inZ, posZ, sizeof(float) * N, hipMemcpyHostToDevice);

		dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
		dim3 gridDim(ceil((double)cube / blockDim.x), ceil((double)cube / blockDim.y), ceil((double)cube / blockDim.z));

		setInitialPos << < gridDim, blockDim >> > (d_inX, d_inY, d_inZ, d_outX, d_outY, d_outZ, N, cube, cube, cube);

		hipMemcpy(initialPosX, d_outX, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipMemcpy(initialPosY, d_outY, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipMemcpy(initialPosZ, d_outZ, sizeof(float) * N, hipMemcpyDeviceToHost);

		hipFree(d_outX);
		hipFree(d_outY);
		hipFree(d_outZ);

		hipFree(d_inX);
		hipFree(d_inY);
		hipFree(d_inZ);

	}


	DllExport const void   cubeMovement(float* posX, float* posY, float* posZ, float* initialPosX, float* initialPosY, float* initialPosZ, int N, int cube, float ciclos) {


		float tau = M_PI * 2;
		float sinFunction = std::sin(tau * ciclos);
		float movement = (sinFunction / 2) + 0.5f;
		/**
		FILE* f = fopen("log_cuda.txt", "a");
		fprintf(f, "movement: %f , sin : %f  \n", movement, sinFunction);
		fprintf(f, "hipMemcpy. posX[0] = %f, posY[0] = %f , posZ[0] = %f\n", posX[0], posY[0], posZ[0]);
		fprintf(f, "hipMemcpy. INiPosX[0] = %f, iniPosY[0] = %f , iniPosZ[0] = %f\n", initialPosX[0], initialPosY[0], initialPosZ[0]);
		fclose(f);
		/**/
		const int THREAD_SIZE = cube;
		const int BLOCK_SIZE = floor(cbrt(1024));

		float* d_X;
		float* d_Y;
		float* d_Z;

		float* initialX;
		float* initialY;
		float* initialZ;

		hipMalloc(&d_X, sizeof(float) * N);
		hipMalloc(&d_Y, sizeof(float) * N);
		hipMalloc(&d_Z, sizeof(float) * N);

		hipMalloc(&initialX, sizeof(float) * N);
		hipMalloc(&initialY, sizeof(float) * N);
		hipMalloc(&initialZ, sizeof(float) * N);

		hipMemcpy(initialX, initialPosX, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(initialY, initialPosY, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(initialZ, initialPosZ, sizeof(float) * N, hipMemcpyHostToDevice);

		/**/
		dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
		dim3 gridDim(ceil((double)cube / blockDim.x), ceil((double)cube / blockDim.y), ceil((double)cube / blockDim.z));

		moveParticles << <gridDim, blockDim >> > (d_X, d_Y, d_Z, initialX, initialY, initialZ, movement, N, cube, cube, cube);
		hipDeviceSynchronize();


		hipMemcpy(posX, d_X, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipMemcpy(posY, d_Y, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipMemcpy(posZ, d_Z, sizeof(float) * N, hipMemcpyDeviceToHost);
		/**
		f = fopen("log_cuda.txt", "a");
		fprintf(f, "hipMemcpy. posX[0] = %f, posY[0] = %f , posZ[0] = %f\n", posX[0], posY[0], posZ[0]);
		fclose(f);
		/**/
		hipFree(d_X);
		hipFree(d_Y);
		hipFree(d_Z);

		hipFree(initialX);
		hipFree(initialY);
		hipFree(initialZ);

	}


}